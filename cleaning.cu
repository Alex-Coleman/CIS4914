#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <thrust/sort.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/count.h>
#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <thrust/pair.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <windows.h>
#include <math.h>
#include "gpuclean.h"
#pragma warning(disable:4503)

#define LANEID_WIDTH 6
#define ZONEID_WIDTH 6
#define TIMESTAMP_WIDTH 22
#define SPEED_WIDTH 4
#define FLOW_WIDTH 4
#define OCCUPANCY_WIDTH 4
#define QUALITY_WIDTH 1

using namespace std;

int main(int argc, char *argv[]) {
    //Set clock
    clock_t start = clock();
    
    //Check arguments
    if (argc != 3) {
        cout << "usage: " << argv[0] << " <lane_detector_inventory> <filename>\n";
        return 0;
    }
    
    //Load Lane Detector Inventory onto GPU
    //End result of this is integer vectors for laneids and zoneids from detector inventory
    HANDLE detectorFile = CreateFileA(argv[1], GENERIC_READ, FILE_SHARE_READ, NULL, OPEN_EXISTING, FILE_ATTRIBUTE_NORMAL | FILE_FLAG_SEQUENTIAL_SCAN, NULL);
    assert(detectorFile != INVALID_HANDLE_VALUE);
    
    HANDLE detectorMap = CreateFileMapping(detectorFile, NULL, PAGE_READONLY, 0, 0, NULL);
    assert(detectorMap != INVALID_HANDLE_VALUE);
 
    LPVOID detectorMapView = MapViewOfFile(detectorMap, FILE_MAP_READ, 0, 0, 0);
    assert(detectorMapView != NULL);
    
    int detectorSize = GetFileSize(detectorFile, NULL);
    char *detectorMapViewChar = (char *)detectorMapView;
    thrust::device_vector<char> detectorCopy(detectorSize);
    thrust::copy(detectorMapViewChar, detectorMapViewChar+detectorSize, detectorCopy.begin());
    cout << "Lane detector inventory loaded onto gpu: " << ((clock() - start)/(double)CLOCKS_PER_SEC) << '\n';
    
    int detector_linecnt = thrust::count(detectorCopy.begin(), detectorCopy.end(), '\n');
    thrust::device_vector<int> detector_linebreaks(detector_linecnt);
    thrust::counting_iterator<int> begin(0);
    thrust::copy_if(begin, begin + detectorSize, detectorCopy.begin(), detector_linebreaks.begin(), line_break());
    
    thrust::device_vector<int> detector_num_columns(1);
    detector_num_columns[0] = 2;
    thrust::device_vector<int> detector_width(detector_num_columns[0]);
    detector_width[0] = LANEID_WIDTH;
    detector_width[1] = ZONEID_WIDTH;
    
    thrust::device_vector<char> detector_laneid(detector_linecnt * detector_width[0]);
    thrust::fill(detector_laneid.begin(), detector_laneid.end(), 0);
    thrust::device_vector<char> detector_zoneid(detector_linecnt * detector_width[1]);
    thrust::fill(detector_zoneid.begin(), detector_zoneid.end(), 0);
    
    thrust::device_vector<char *> detector_columns(2);
    detector_columns[0] = thrust::raw_pointer_cast(detector_laneid.data());
    detector_columns[1] = thrust::raw_pointer_cast(detector_zoneid.data());
    column_split detector_split((char *)thrust::raw_pointer_cast(detectorCopy.data()), (int *)thrust::raw_pointer_cast(detector_linebreaks.data()), (char **)thrust::raw_pointer_cast(detector_columns.data()), (int *)thrust::raw_pointer_cast(detector_width.data()), (int *)thrust::raw_pointer_cast(detector_num_columns.data()));
    thrust::for_each(begin, begin + detector_linecnt, detector_split);
    
    thrust::device_vector<int> unique_laneid(detector_linecnt);
    gpu_atoi get_laneid((char *)thrust::raw_pointer_cast(detector_laneid.data()), (int *)thrust::raw_pointer_cast(unique_laneid.data()), (int *)thrust::raw_pointer_cast(detector_width.data()));
    thrust::for_each(begin, begin + detector_linecnt, get_laneid);
    thrust::device_vector<int> unique_zoneid(detector_linecnt);
    gpu_atoi get_zoneid((char *)thrust::raw_pointer_cast(detector_zoneid.data()), (int *)thrust::raw_pointer_cast(unique_zoneid.data()), (int *)thrust::raw_pointer_cast(&detector_width[1]));
    thrust::for_each(begin, begin + detector_linecnt, get_zoneid);
    cout << "Lane detector inventory parsed: " << ((clock() - start)/(double)CLOCKS_PER_SEC) << '\n';

    
    //Now we load the actual file to be cleaned
    //Windows memory mapping
    HANDLE file = CreateFileA(argv[2], GENERIC_READ, FILE_SHARE_READ, NULL, OPEN_EXISTING, FILE_ATTRIBUTE_NORMAL | FILE_FLAG_SEQUENTIAL_SCAN, NULL);
    assert(file != INVALID_HANDLE_VALUE);
    
    HANDLE fileMap = CreateFileMapping(file, NULL, PAGE_READONLY, 0, 0, NULL);
    assert(fileMap != INVALID_HANDLE_VALUE);
 
    LPVOID fileMapView = MapViewOfFile(fileMap, FILE_MAP_READ, 0, 0, 0);
    assert(fileMapView != NULL);

    //Copy file to GPU
    long fileSize = GetFileSize(file, NULL);
    char *fileMapViewChar = (char *)fileMapView;
    thrust::device_vector<char> fileCopy(fileSize);
    thrust::copy(fileMapViewChar, fileMapViewChar+fileSize, fileCopy.begin());
    cout << "File to be cleaned loaded onto gpu: " << ((clock() - start)/(double)CLOCKS_PER_SEC) << '\n';

    
    //Measure linebreaks, store their location in device vector
    int linecnt = thrust::count(fileCopy.begin(), fileCopy.end(), '\n');
    thrust::device_vector<int> linebreaks(linecnt);
    thrust::copy_if(begin, begin + fileSize, fileCopy.begin(), linebreaks.begin(), line_break());
    
    //Store column widths in device vector
    thrust::device_vector<int> num_columns(1);
    num_columns[0] = 6;
    thrust::device_vector<int> column_width(num_columns[0]);
    column_width[0] = LANEID_WIDTH;
    column_width[1] = TIMESTAMP_WIDTH;
    column_width[2] = SPEED_WIDTH;
    column_width[3] = FLOW_WIDTH;
    column_width[4] = OCCUPANCY_WIDTH;
    column_width[5] = QUALITY_WIDTH;
    
    //Create vectors for each column
    thrust::device_vector<char> laneid_text(linecnt*column_width[0]);
    thrust::fill(laneid_text.begin(), laneid_text.end(), 0);
    thrust::device_vector<char> timestamp_text(linecnt*column_width[1]);
    thrust::fill(timestamp_text.begin(), timestamp_text.end(), 0);
    thrust::device_vector<char> speed_text(linecnt*column_width[2]);
    thrust::fill(speed_text.begin(), speed_text.end(), 0);
    thrust::device_vector<char> flow_text(linecnt*column_width[3]);
    thrust::fill(flow_text.begin(), flow_text.end(), 0);
    thrust::device_vector<char> occupancy_text(linecnt*column_width[4]);
    thrust::fill(occupancy_text.begin(), occupancy_text.end(), 0);
    thrust::device_vector<char> quality_text(linecnt*column_width[5]);
    thrust::fill(quality_text.begin(), quality_text.end(), 0);
    
    //Vector to store all of the columns
    thrust::device_vector<char *> columns(6);
    columns[0] = thrust::raw_pointer_cast(laneid_text.data());
    columns[1] = thrust::raw_pointer_cast(timestamp_text.data());
    columns[2] = thrust::raw_pointer_cast(speed_text.data());
    columns[3] = thrust::raw_pointer_cast(flow_text.data());
    columns[4] = thrust::raw_pointer_cast(occupancy_text.data());
    columns[5] = thrust::raw_pointer_cast(quality_text.data());
    
    //Split the text into 6 columns
    column_split splitter((char *)thrust::raw_pointer_cast(fileCopy.data()), (int *)thrust::raw_pointer_cast(linebreaks.data()), (char **)thrust::raw_pointer_cast(columns.data()), (int *)thrust::raw_pointer_cast(column_width.data()), (int *)thrust::raw_pointer_cast(num_columns.data()));
    thrust::for_each(begin, begin + linecnt, splitter);
    
    //We need to convert each vector to the appropriate type
    //Laneid
    thrust::device_vector<int> laneid(linecnt);
    gpu_atoi laneid_toint((char *)thrust::raw_pointer_cast(laneid_text.data()), (int *)thrust::raw_pointer_cast(laneid.data()), (int *)thrust::raw_pointer_cast(column_width.data()));
    thrust::for_each(begin, begin + linecnt, laneid_toint);
    
    //Flow
    thrust::device_vector<int> flow(linecnt);
    gpu_atoi flow_toint((char *)thrust::raw_pointer_cast(flow_text.data()), (int *)thrust::raw_pointer_cast(flow.data()), (int *)thrust::raw_pointer_cast(&column_width[3]));
    thrust::for_each(begin, begin + linecnt, flow_toint);
    cout << "File to be cleaned parsed: " << ((clock() - start)/(double)CLOCKS_PER_SEC) << '\n';
    
    
    //Now we want to figure out the appropriate zoneid for each entry
    thrust::device_vector<int> zoneid(linecnt);
    thrust::device_vector<int> device_detector_linecnt(1);
    device_detector_linecnt[0] = detector_linecnt;
    column_search assign_zoneid((int *)thrust::raw_pointer_cast(unique_laneid.data()), (int *)thrust::raw_pointer_cast(unique_zoneid.data()), (int *)thrust::raw_pointer_cast(laneid.data()), (int *)thrust::raw_pointer_cast(zoneid.data()), (int *)thrust::raw_pointer_cast(device_detector_linecnt.data()));
    thrust::for_each(begin, begin + linecnt, assign_zoneid);
    cout << "Zoneid matched to each entry: " << ((clock() - start)/(double)CLOCKS_PER_SEC) << '\n';

    
    //CLEAN
    //Check bounds on flow values
    thrust::device_vector<char> flow_valid(linecnt);
    thrust::fill(flow_valid.begin(), flow_valid.end(), '0');
    thrust::device_vector<int> flow_bounds(2);
    flow_bounds[0] = 0;
    flow_bounds[1] = 200;
    check_bounds check_flow((int *)thrust::raw_pointer_cast(flow.data()), (char *)thrust::raw_pointer_cast(flow_valid.data()), (int *)thrust::raw_pointer_cast(flow_bounds.data()));
    thrust::for_each(begin, begin + linecnt, check_flow);
    
    summary_stats_unary_op  unary_op((int *)thrust::raw_pointer_cast(flow.data()), (char *)thrust::raw_pointer_cast(flow_valid.data()));
    summary_stats_binary_op binary_op;
    summary_stats_data      init;
    
    init.initialize();
    
    summary_stats_data result = thrust::transform_reduce(begin, begin + linecnt, unary_op, init, binary_op);
    
    /*cout << "flow: " << flow[471] << "\n";
    cout << "valid: " << flow_valid[471] << "\n";
    
    cout << "N: " << result.n << "\n";
    cout << "M2: " << result.M2 << "\n";
    cout << "Mean: " << result.mean << "\n";*/
    cout << "Standard Deviation: " << sqrt(result.variance_n()) << "\n";
    cout << "Bounds checked, standard deviation calculated: " << ((clock() - start)/(double)CLOCKS_PER_SEC) << '\n';

    
    //Create index for entries
    thrust::device_vector<int> index(linecnt);
    index_filler fill_index((int *)thrust::raw_pointer_cast(index.data()));
    thrust::for_each(begin, begin+linecnt, fill_index);
    
    //Sort zoneid and index
    thrust::stable_sort_by_key(zoneid.begin(), zoneid.end(), index.begin());
    
    //Clean by checking standard deviation
    thrust::device_vector<int> device_linecnt(1);
    thrust::device_vector<int> new_flow(linecnt);
    device_linecnt[0] = linecnt;
    thrust::device_vector<float> global_std(1);
    global_std[0] = sqrt(result.variance_n());
    
    std_clean cleaner((int *)thrust::raw_pointer_cast(zoneid.data()), (int *)thrust::raw_pointer_cast(index.data()), (int *)thrust::raw_pointer_cast(flow.data()), (int *)thrust::raw_pointer_cast(new_flow.data()), (char *)thrust::raw_pointer_cast(flow_valid.data()), (int *)thrust::raw_pointer_cast(device_linecnt.data()), (float *)thrust::raw_pointer_cast(global_std.data()));
    thrust::for_each(begin, begin+linecnt, cleaner);
    cout << "Data cleaned: " << ((clock() - start)/(double)CLOCKS_PER_SEC) << '\n';

    
    //Convert values back to char
    //Laneid
    thrust::fill(laneid_text.begin(), laneid_text.end(), 0);
    gpu_itoa laneid_tochar((int *)thrust::raw_pointer_cast(laneid.data()), (char *)thrust::raw_pointer_cast(laneid_text.data()), (int *)thrust::raw_pointer_cast(column_width.data()));
    thrust::for_each(begin, begin + linecnt, laneid_tochar);
    
    //Flow
    thrust::fill(flow_text.begin(), flow_text.end(), 0);
    gpu_itoa flow_tochar((int *)thrust::raw_pointer_cast(flow.data()), (char *)thrust::raw_pointer_cast(flow_text.data()), (int *)thrust::raw_pointer_cast(&column_width[3]));
    thrust::for_each(begin, begin + linecnt, flow_tochar);
    
    //New Flow
    thrust::device_vector<char> new_flow_text(linecnt*column_width[3]);
    thrust::fill(new_flow_text.begin(), new_flow_text.end(), 0);
    gpu_itoa new_flow_tochar((int *)thrust::raw_pointer_cast(new_flow.data()), (char *)thrust::raw_pointer_cast(new_flow_text.data()), (int *)thrust::raw_pointer_cast(&column_width[3]));
    thrust::for_each(begin, begin + linecnt, new_flow_tochar);
    //thrust::host_vector<char> flow_host(linecnt*column_width[3]);
    //thrust::copy(flow_text.begin(), flow_text.end(), flow_host.begin());
    //cout << "Laneid and flow moved to host: " << ((clock() - start)/(double)CLOCKS_PER_SEC) << '\n';
    
    //Format output on GPU
    //Output includes laneid, previous flow, new flow, and validity, separated by commas
    thrust::device_vector<int> output_num_columns(1);
    output_num_columns[0] = 4;
    
    thrust::device_vector<char *> output_columns(output_num_columns[0]);
    output_columns[0] = thrust::raw_pointer_cast(laneid_text.data());
    output_columns[1] = thrust::raw_pointer_cast(flow_text.data());
    output_columns[2] = thrust::raw_pointer_cast(new_flow_text.data());
    output_columns[3] = thrust::raw_pointer_cast(flow_valid.data());
    
    thrust::device_vector<int> output_column_width(output_num_columns[0]);
    output_column_width[0] = column_width[0];
    output_column_width[1] = column_width[3];
    output_column_width[2] = column_width[3];
    output_column_width[3] = column_width[5];
    
    thrust::device_vector<int> output_size(1);
    output_size[0] = output_column_width[0] + output_column_width[1] + output_column_width[2] + output_column_width[3] + output_column_width.size();
    thrust::device_vector<char> output(linecnt * output_size[0]);
    thrust::fill(output.begin(), output.end(), ' ');
    gpu_output format_output((char **)thrust::raw_pointer_cast(output_columns.data()), (char *)thrust::raw_pointer_cast(output.data()), (int *)thrust::raw_pointer_cast(output_column_width.data()), (int *)thrust::raw_pointer_cast(output_size.data()), (int *)thrust::raw_pointer_cast(output_num_columns.data()));
    thrust::for_each(begin, begin + linecnt, format_output);
    thrust::device_vector<char> final_output(output.size());
    thrust::copy_if(output.begin(), output.end(), final_output.begin(), white_space());
    thrust::device_vector<char>::iterator output_end = thrust::find(final_output.begin(), final_output.end(), NULL);

    
    //Output cleaned data
    assert(SetCurrentDirectory("Cleaned") != 0);
    HANDLE outputFile = CreateFileA(argv[2], GENERIC_WRITE | GENERIC_READ, FILE_SHARE_WRITE, NULL, CREATE_ALWAYS, FILE_ATTRIBUTE_NORMAL | FILE_FLAG_SEQUENTIAL_SCAN, NULL);
    assert(outputFile != INVALID_HANDLE_VALUE);
    
    HANDLE outputMap = CreateFileMapping(outputFile, NULL, PAGE_READWRITE, 0, output_end - final_output.begin(), NULL);
    if(outputMap == NULL)
        cout << GetLastError();        
 
    LPVOID outputMapView = MapViewOfFile(outputMap, FILE_MAP_WRITE, 0, 0, 0);
    
    if (outputMapView == NULL)
        cout << GetLastError();
    
    char *outputMapViewChar = (char *)outputMapView;
    thrust::copy(final_output.begin(), output_end, outputMapViewChar);

    cout << "Output written: " << ((clock() - start)/(double)CLOCKS_PER_SEC) << '\n';

    return 0;
}