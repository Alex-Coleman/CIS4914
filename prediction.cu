#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <thrust/sort.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/count.h>
#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <thrust/pair.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <windows.h>
#include <math.h>
#include "gpupredict.h"
#include <hiprand/hiprand_kernel.h>
#include <thrust/random.h>
#pragma warning(disable:4503)

#define NUMBER_OF_COLUMNS 4
#define TIMESTAMP_WIDTH 22
#define EVENT_WIDTH 32
#define LATITUDE_WIDTH 11
#define LONGITUDE_WIDTH 11
#define NUMBER_OF_BOUNDING_BOXES 300
/*
#define FLOW_WIDTH 4
#define OCCUPANCY_WIDTH 4
#define QUALITY_WIDTH 1
*/
using namespace std;

int main(int argc, char *argv[]) {
    //Set clock
    clock_t start = clock();
    
    //Check arguments
    if (argc != 2) {
        cout << "usage: " << argv[0] << " <events_train>\n";
        return 0;
    }
    
    //Load Lane Detector Inventory onto GPU
    //End result of this is integer vectors for laneids and zoneids from detector inventory
    /*HANDLE detectorFile = CreateFileA(argv[1], GENERIC_READ, FILE_SHARE_READ, NULL, OPEN_EXISTING, FILE_ATTRIBUTE_NORMAL | FILE_FLAG_SEQUENTIAL_SCAN, NULL);
    assert(detectorFile != INVALID_HANDLE_VALUE);
    
    HANDLE detectorMap = CreateFileMapping(detectorFile, NULL, PAGE_READONLY, 0, 0, NULL);
    assert(detectorMap != INVALID_HANDLE_VALUE);
 
    LPVOID detectorMapView = MapViewOfFile(detectorMap, FILE_MAP_READ, 0, 0, 0);
    assert(detectorMapView != NULL);
    
    int detectorSize = GetFileSize(detectorFile, NULL);
    char *detectorMapViewChar = (char *)detectorMapView;
    thrust::device_vector<char> detectorCopy(detectorSize);
    thrust::copy(detectorMapViewChar, detectorMapViewChar+detectorSize, detectorCopy.begin());
    
    int detector_linecnt = thrust::count(detectorCopy.begin(), detectorCopy.end(), '\n');
    thrust::device_vector<int> detector_linebreaks(detector_linecnt);
    thrust::counting_iterator<int> begin(0);
    thrust::copy_if(begin, begin + detectorSize, detectorCopy.begin(), detector_linebreaks.begin(), line_break());
    
    thrust::device_vector<int> detector_num_columns(1);
    detector_num_columns[0] = 2;
    thrust::device_vector<int> detector_width(detector_num_columns[0]);
    detector_width[0] = LANEID_WIDTH;
    detector_width[1] = ZONEID_WIDTH;
    
    thrust::device_vector<char> detector_laneid(detector_linecnt * detector_width[0]);
    thrust::fill(detector_laneid.begin(), detector_laneid.end(), 0);
    thrust::device_vector<char> detector_zoneid(detector_linecnt * detector_width[1]);
    thrust::fill(detector_zoneid.begin(), detector_zoneid.end(), 0);
    
    thrust::device_vector<char *> detector_columns(2);
    detector_columns[0] = thrust::raw_pointer_cast(detector_laneid.data());
    detector_columns[1] = thrust::raw_pointer_cast(detector_zoneid.data());
    column_split detector_split((char *)thrust::raw_pointer_cast(detectorCopy.data()), (int *)thrust::raw_pointer_cast(detector_linebreaks.data()), (char **)thrust::raw_pointer_cast(detector_columns.data()), (int *)thrust::raw_pointer_cast(detector_width.data()), (int *)thrust::raw_pointer_cast(detector_num_columns.data()));
    thrust::for_each(begin, begin + detector_linecnt, detector_split);
    
    thrust::device_vector<int> unique_laneid(detector_linecnt);
    gpu_atoi get_laneid((char *)thrust::raw_pointer_cast(detector_laneid.data()), (int *)thrust::raw_pointer_cast(unique_laneid.data()), (int *)thrust::raw_pointer_cast(detector_width.data()));
    thrust::for_each(begin, begin + detector_linecnt, get_laneid);
    thrust::device_vector<int> unique_zoneid(detector_linecnt);
    gpu_atoi get_zoneid((char *)thrust::raw_pointer_cast(detector_zoneid.data()), (int *)thrust::raw_pointer_cast(unique_zoneid.data()), (int *)thrust::raw_pointer_cast(&detector_width[1]));
    thrust::for_each(begin, begin + detector_linecnt, get_zoneid);*/
    
    //Now we load the actual file to be cleaned
    //Windows memory mapping
    HANDLE file = CreateFileA(argv[1], GENERIC_READ, FILE_SHARE_READ, NULL, OPEN_EXISTING, FILE_ATTRIBUTE_NORMAL | FILE_FLAG_SEQUENTIAL_SCAN, NULL);
    assert(file != INVALID_HANDLE_VALUE);
    
    HANDLE fileMap = CreateFileMapping(file, NULL, PAGE_READONLY, 0, 0, NULL);
    assert(fileMap != INVALID_HANDLE_VALUE);
 
    LPVOID fileMapView = MapViewOfFile(fileMap, FILE_MAP_READ, 0, 0, 0);
    assert(fileMapView != NULL);

    //Copy file to GPU
    int fileSize = GetFileSize(file, NULL);
    char *fileMapViewChar = (char *)fileMapView;
    thrust::device_vector<char> fileCopy(fileSize);
    thrust::copy(fileMapViewChar, fileMapViewChar+fileSize, fileCopy.begin());
    
    //Measure linebreaks, store their location in device vector
    int linecnt = thrust::count(fileCopy.begin(), fileCopy.end(), '\n');
    thrust::device_vector<int> linebreaks(linecnt);
    thrust::counting_iterator<int> begin(0);
    thrust::copy_if(begin, begin + fileSize, fileCopy.begin(), linebreaks.begin(), line_break());
    
    //Store column widths in device vector
    thrust::device_vector<int> num_columns(1);
    num_columns[0] = NUMBER_OF_COLUMNS;
    thrust::device_vector<int> column_width(num_columns[0]);
    column_width[0] = TIMESTAMP_WIDTH;
    column_width[1] = EVENT_WIDTH;
    column_width[2] = LATITUDE_WIDTH;
    column_width[3] = LONGITUDE_WIDTH;
    
    //Create vectors for each column
    thrust::device_vector<char> tstamp(linecnt*column_width[0]);
    thrust::fill(tstamp.begin(), tstamp.end(), 0);
    thrust::device_vector<char> event(linecnt*column_width[1]);
    thrust::fill(event.begin(), event.end(), 0);
    thrust::device_vector<char> latitude_text(linecnt*column_width[2]);
    thrust::fill(latitude_text.begin(), latitude_text.end(), 0);
    thrust::device_vector<char> longitude_text(linecnt*column_width[3]);
    thrust::fill(longitude_text.begin(), longitude_text.end(), 0);
    
    //Vector to store all of the columns
    thrust::device_vector<char *> columns(num_columns[0]);
    columns[0] = thrust::raw_pointer_cast(tstamp.data());
    columns[1] = thrust::raw_pointer_cast(event.data());
    columns[2] = thrust::raw_pointer_cast(latitude_text.data());
    columns[3] = thrust::raw_pointer_cast(longitude_text.data());
    
    thrust::device_vector<int> column_locations(num_columns[0]);
    column_locations[0] = 4;
    column_locations[1] = 6;
    column_locations[2] = 9;
    column_locations[3] = 10;
    
    //Split the text into 6 columns
    column_split splitter((char *)thrust::raw_pointer_cast(fileCopy.data()), (int *)thrust::raw_pointer_cast(linebreaks.data()), (char **)thrust::raw_pointer_cast(columns.data()), (int *)thrust::raw_pointer_cast(column_width.data()), (int *)thrust::raw_pointer_cast(num_columns.data()), (int *)thrust::raw_pointer_cast(column_locations.data()));
    thrust::for_each(begin, begin + linecnt, splitter);
    
    /*//Convert Latitude and Longitude to floats
    thrust::device_vector<double> latitude(linecnt);
    gpu_atof latitude_tofloat((char *)thrust::raw_pointer_cast(latitude_text.data()), (double *)thrust::raw_pointer_cast(latitude.data()), (int *)thrust::raw_pointer_cast(&column_width[5]));
    thrust::for_each(begin, begin + linecnt, latitude_tofloat);  
    thrust::device_vector<double> longitude(linecnt);
    gpu_atof longitude_tofloat((char *)thrust::raw_pointer_cast(longitude_text.data()), (double *)thrust::raw_pointer_cast(longitude.data()), (int *)thrust::raw_pointer_cast(&column_width[6]));
    thrust::for_each(begin, begin + linecnt, longitude_tofloat);
    
    //Get the month from the timestamp, since that's all we need
    thrust::device_vector<int> month(linecnt);
    get_month get_months((char *)thrust::raw_pointer_cast(tstamp.data()), (int *)thrust::raw_pointer_cast(month.data()));
    thrust::for_each(begin, begin + linecnt, get_months);
    //thrust::copy(created_tstamp.begin(), created_tstamp.end(), ostream_iterator<char>(cout));
    
    thrust::device_vector<double> bb_min_latitude(NUMBER_OF_BOUNDING_BOXES);
    thrust::device_vector<double> bb_max_latitude(NUMBER_OF_BOUNDING_BOXES);
    thrust::device_vector<double> bb_min_longitude(NUMBER_OF_BOUNDING_BOXES);
    thrust::device_vector<double> bb_max_longitude(NUMBER_OF_BOUNDING_BOXES);
    thrust::device_vector<int> bb_month(NUMBER_OF_BOUNDING_BOXES);*/
    
    //thrust::default_random_engine rng;
    //thrust::uniform_real_distribution<double> dist;
    //cout << dist(rng) << "\n";
    //cout << dist(rng) << "\n";
    //rng.discard()
    
    //create_bb bounding_boxes((double *)thrust::raw_pointer_cast(bb_min_latitude.data()), (double *)thrust::raw_pointer_cast(bb_max_latitude.data()), (double *)thrust::raw_pointer_cast(bb_min_longitude.data()), (double *)thrust::raw_pointer_cast(bb_max_longitude.data()), (int *)thrust::raw_pointer_cast(bb_month.data()));
    //thrust::for_each(begin, begin + NUMBER_OF_BOUNDING_BOXES, bounding_boxes);
    
    
    //thrust::sort(longitude.begin(), longitude.end());
    //ofstream output("output2.csv");
    //thrust::copy(longitude.begin(), longitude.end(), ostream_iterator<double>(output, "\n"));
    //thrust::sort(longitude.begin(), longitude.end());
    //cout << latitude[0] << '\n';
    //cout << latitude[linecnt-1] << '\n';
    //cout << longitude[0] << '\n';
    //cout << longitude[linecnt - 1] << '\n';
    
    //cout.precision(column_width[5]);
    //thrust::copy(latitude.begin(), latitude.begin() + 100, ostream_iterator<double>(cout, "\n"));
    
    /*
    //We need to convert each vector to the appropriate type
    //Laneid
    thrust::device_vector<int> laneid(linecnt);
    gpu_atoi laneid_toint((char *)thrust::raw_pointer_cast(laneid_text.data()), (int *)thrust::raw_pointer_cast(laneid.data()), (int *)thrust::raw_pointer_cast(column_width.data()));
    thrust::for_each(begin, begin + linecnt, laneid_toint);
    
    //Flow
    thrust::device_vector<int> flow(linecnt);
    gpu_atoi flow_toint((char *)thrust::raw_pointer_cast(flow_text.data()), (int *)thrust::raw_pointer_cast(flow.data()), (int *)thrust::raw_pointer_cast(&column_width[3]));
    thrust::for_each(begin, begin + linecnt, flow_toint);
    cout << "Lane detector inventory and sensor data parsed: " << ((clock() - start)/(double)CLOCKS_PER_SEC) << '\n';
    
    //Now we want to figure out the appropriate zoneid for each entry
    thrust::device_vector<int> zoneid(linecnt);
    thrust::device_vector<int> device_detector_linecnt(1);
    device_detector_linecnt[0] = detector_linecnt;
    column_search assign_zoneid((int *)thrust::raw_pointer_cast(unique_laneid.data()), (int *)thrust::raw_pointer_cast(unique_zoneid.data()), (int *)thrust::raw_pointer_cast(laneid.data()), (int *)thrust::raw_pointer_cast(zoneid.data()), (int *)thrust::raw_pointer_cast(device_detector_linecnt.data()));
    thrust::for_each(begin, begin + linecnt, assign_zoneid);
    
    //CLEAN
    //Check bounds on flow values
    thrust::device_vector<char> flow_valid(linecnt);
    thrust::fill(flow_valid.begin(), flow_valid.end(), '0');
    thrust::device_vector<int> flow_bounds(2);
    flow_bounds[0] = 0;
    flow_bounds[1] = 100;
    check_bounds check_flow((int *)thrust::raw_pointer_cast(flow.data()), (char *)thrust::raw_pointer_cast(flow_valid.data()), (int *)thrust::raw_pointer_cast(flow_bounds.data()));
    thrust::for_each(begin, begin + linecnt, check_flow);
    
    //Create index for entries
    thrust::device_vector<int> index(linecnt);
    index_filler fill_index((int *)thrust::raw_pointer_cast(index.data()));
    thrust::for_each(begin, begin+linecnt, fill_index);
    
    //Sort zoneid and index
    thrust::stable_sort_by_key(zoneid.begin(), zoneid.end(), index.begin());
    
    //Clean by checking standard deviation
    thrust::device_vector<int> device_linecnt(1);
    thrust::device_vector<int> new_flow(linecnt);
    device_linecnt[0] = linecnt;
    std_clean cleaner((int *)thrust::raw_pointer_cast(zoneid.data()), (int *)thrust::raw_pointer_cast(index.data()), (int *)thrust::raw_pointer_cast(flow.data()), (int *)thrust::raw_pointer_cast(new_flow.data()), (char *)thrust::raw_pointer_cast(flow_valid.data()), (int *)thrust::raw_pointer_cast(device_linecnt.data()));
    thrust::for_each(begin, begin+linecnt, cleaner);
    
    //Convert values back to char
    //Laneid
    thrust::fill(laneid_text.begin(), laneid_text.end(), 0);
    gpu_itoa laneid_tochar((int *)thrust::raw_pointer_cast(laneid.data()), (char *)thrust::raw_pointer_cast(laneid_text.data()), (int *)thrust::raw_pointer_cast(column_width.data()));
    thrust::for_each(begin, begin + linecnt, laneid_tochar);
    
    //Flow
    thrust::fill(flow_text.begin(), flow_text.end(), 0);
    gpu_itoa flow_tochar((int *)thrust::raw_pointer_cast(flow.data()), (char *)thrust::raw_pointer_cast(flow_text.data()), (int *)thrust::raw_pointer_cast(&column_width[3]));
    thrust::for_each(begin, begin + linecnt, flow_tochar);
    
    //New Flow
    thrust::device_vector<char> new_flow_text(linecnt*column_width[3]);
    thrust::fill(new_flow_text.begin(), new_flow_text.end(), 0);
    gpu_itoa new_flow_tochar((int *)thrust::raw_pointer_cast(new_flow.data()), (char *)thrust::raw_pointer_cast(new_flow_text.data()), (int *)thrust::raw_pointer_cast(&column_width[3]));
    thrust::for_each(begin, begin + linecnt, new_flow_tochar);
    //thrust::host_vector<char> flow_host(linecnt*column_width[3]);
    //thrust::copy(flow_text.begin(), flow_text.end(), flow_host.begin());
    //cout << "Laneid and flow moved to host: " << ((clock() - start)/(double)CLOCKS_PER_SEC) << '\n';
    
    //Format output on GPU
    //Output includes laneid, previous flow, new flow, and validity, separated by commas
    thrust::device_vector<int> output_num_columns(1);
    output_num_columns[0] = 4;
    
    thrust::device_vector<char *> output_columns(output_num_columns[0]);
    output_columns[0] = thrust::raw_pointer_cast(laneid_text.data());
    output_columns[1] = thrust::raw_pointer_cast(flow_text.data());
    output_columns[2] = thrust::raw_pointer_cast(new_flow_text.data());
    output_columns[3] = thrust::raw_pointer_cast(flow_valid.data());
    
    thrust::device_vector<int> output_column_width(output_num_columns[0]);
    output_column_width[0] = column_width[0];
    output_column_width[1] = column_width[3];
    output_column_width[2] = column_width[3];
    output_column_width[3] = column_width[5];
    
    thrust::device_vector<int> output_size(1);
    output_size[0] = output_column_width[0] + output_column_width[1] + output_column_width[2] + output_column_width[3] + output_column_width.size();
    thrust::device_vector<char> output(linecnt * output_size[0]);
    thrust::fill(output.begin(), output.end(), ' ');
    gpu_output format_output((char **)thrust::raw_pointer_cast(output_columns.data()), (char *)thrust::raw_pointer_cast(output.data()), (int *)thrust::raw_pointer_cast(output_column_width.data()), (int *)thrust::raw_pointer_cast(output_size.data()), (int *)thrust::raw_pointer_cast(output_num_columns.data()));
    thrust::for_each(begin, begin + linecnt, format_output);
    
    //Output cleaned data
    assert(SetCurrentDirectory("Cleaned") != 0);
    HANDLE outputFile = CreateFileA(argv[2], GENERIC_WRITE | GENERIC_READ, FILE_SHARE_WRITE, NULL, CREATE_ALWAYS, FILE_ATTRIBUTE_NORMAL | FILE_FLAG_SEQUENTIAL_SCAN, NULL);
    assert(outputFile != INVALID_HANDLE_VALUE);
    
    HANDLE outputMap = CreateFileMapping(outputFile, NULL, PAGE_READWRITE, 0, output.size(), NULL);
    if(outputMap == NULL)
        cout << GetLastError();        
 
    LPVOID outputMapView = MapViewOfFile(outputMap, FILE_MAP_WRITE, 0, 0, 0);
    
    if (outputMapView == NULL)
        cout << GetLastError();
    
    char *outputMapViewChar = (char *)outputMapView;
    thrust::copy(output.begin(), output.end(), outputMapViewChar);
    //ofstream output(argv[1]);
    //thrust::copy(laneid_text.begin(), laneid_text.end(), ostream_iterator<char>(output));
    cout << "Output written: " << ((clock() - start)/(double)CLOCKS_PER_SEC) << '\n';
    */
    return 0;
}