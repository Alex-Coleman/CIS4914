#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <thrust/sort.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/count.h>
#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <thrust/pair.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <windows.h>
#include <math.h>
#include "gpupredict.h"
#include <hiprand/hiprand_kernel.h>
#include <thrust/random.h>
#pragma warning(disable:4503)

#define NUMBER_OF_COLUMNS 4
#define TIMESTAMP_WIDTH 22
#define EVENT_WIDTH 32
#define LATITUDE_WIDTH 11
#define LONGITUDE_WIDTH 11
#define NUMBER_OF_BOUNDING_BOXES 300
#define NUMBER_OF_OUTPUT_COLUMNS 6
#define MONTH_WIDTH 2
#define EVENT_COUNT_WIDTH 6

using namespace std;

int main(int argc, char *argv[]) {
    //Set clock
    clock_t start = clock();
    
    //Check arguments
    if (argc != 2) {
        cout << "usage: " << argv[0] << " <events_train>\n";
        return 0;
    }
    
    //Now we load the actual file to be cleaned
    //Windows memory mapping
    HANDLE file = CreateFileA(argv[1], GENERIC_READ, FILE_SHARE_READ, NULL, OPEN_EXISTING, FILE_ATTRIBUTE_NORMAL | FILE_FLAG_SEQUENTIAL_SCAN, NULL);
    assert(file != INVALID_HANDLE_VALUE);
    
    HANDLE fileMap = CreateFileMapping(file, NULL, PAGE_READONLY, 0, 0, NULL);
    assert(fileMap != INVALID_HANDLE_VALUE);
 
    LPVOID fileMapView = MapViewOfFile(fileMap, FILE_MAP_READ, 0, 0, 0);
    assert(fileMapView != NULL);

    //Copy file to GPU
    int fileSize = GetFileSize(file, NULL);
    char *fileMapViewChar = (char *)fileMapView;
    thrust::device_vector<char> fileCopy(fileSize);
    thrust::copy(fileMapViewChar, fileMapViewChar+fileSize, fileCopy.begin());
    
    //Measure linebreaks, store their location in device vector
    int linecnt = thrust::count(fileCopy.begin(), fileCopy.end(), '\n');
    thrust::device_vector<int> linebreaks(linecnt);
    thrust::counting_iterator<int> begin(0);
    thrust::copy_if(begin, begin + fileSize, fileCopy.begin(), linebreaks.begin(), line_break());
    
    //Store column widths in device vector
    thrust::device_vector<int> num_columns(1);
    num_columns[0] = NUMBER_OF_COLUMNS;
    thrust::device_vector<int> column_width(num_columns[0]);
    column_width[0] = TIMESTAMP_WIDTH;
    column_width[1] = EVENT_WIDTH;
    column_width[2] = LATITUDE_WIDTH;
    column_width[3] = LONGITUDE_WIDTH;
    
    //Create vectors for each column
    thrust::device_vector<char> tstamp(linecnt*column_width[0]);
    thrust::fill(tstamp.begin(), tstamp.end(), 0);
    thrust::device_vector<char> event(linecnt*column_width[1]);
    thrust::fill(event.begin(), event.end(), 0);
    thrust::device_vector<char> latitude_text(linecnt*column_width[2]);
    thrust::fill(latitude_text.begin(), latitude_text.end(), 0);
    thrust::device_vector<char> longitude_text(linecnt*column_width[3]);
    thrust::fill(longitude_text.begin(), longitude_text.end(), 0);
    
    //Vector to store all of the columns
    thrust::device_vector<char *> columns(num_columns[0]);
    columns[0] = thrust::raw_pointer_cast(tstamp.data());
    columns[1] = thrust::raw_pointer_cast(event.data());
    columns[2] = thrust::raw_pointer_cast(latitude_text.data());
    columns[3] = thrust::raw_pointer_cast(longitude_text.data());
    
    thrust::device_vector<int> column_locations(num_columns[0]);
    column_locations[0] = 4;
    column_locations[1] = 6;
    column_locations[2] = 9;
    column_locations[3] = 10;
    
    //Split the text into 6 columns
    column_split splitter((char *)thrust::raw_pointer_cast(fileCopy.data()), (int *)thrust::raw_pointer_cast(linebreaks.data()), (char **)thrust::raw_pointer_cast(columns.data()), (int *)thrust::raw_pointer_cast(column_width.data()), (int *)thrust::raw_pointer_cast(num_columns.data()), (int *)thrust::raw_pointer_cast(column_locations.data()));
    thrust::for_each(begin, begin + linecnt, splitter);
    
    //Convert Latitude and Longitude to floats
    thrust::device_vector<double> latitude(linecnt);
    gpu_atof latitude_tofloat((char *)thrust::raw_pointer_cast(latitude_text.data()), (double *)thrust::raw_pointer_cast(latitude.data()), (int *)thrust::raw_pointer_cast(&column_width[2]));
    thrust::for_each(begin, begin + linecnt, latitude_tofloat);  
    thrust::device_vector<double> longitude(linecnt);
    gpu_atof longitude_tofloat((char *)thrust::raw_pointer_cast(longitude_text.data()), (double *)thrust::raw_pointer_cast(longitude.data()), (int *)thrust::raw_pointer_cast(&column_width[3]));
    thrust::for_each(begin, begin + linecnt, longitude_tofloat);
    
    //Get the month from the timestamp, since that's all we need
    thrust::device_vector<int> month(linecnt);
    get_month get_months((char *)thrust::raw_pointer_cast(tstamp.data()), (int *)thrust::raw_pointer_cast(month.data()));
    thrust::for_each(begin, begin + linecnt, get_months);
    
    thrust::device_vector<double> bb_min_latitude(NUMBER_OF_BOUNDING_BOXES);
    thrust::device_vector<double> bb_max_latitude(NUMBER_OF_BOUNDING_BOXES);
    thrust::device_vector<double> bb_min_longitude(NUMBER_OF_BOUNDING_BOXES);
    thrust::device_vector<double> bb_max_longitude(NUMBER_OF_BOUNDING_BOXES);
    thrust::device_vector<int> bb_month(NUMBER_OF_BOUNDING_BOXES);
    
    create_bb bounding_boxes((double *)thrust::raw_pointer_cast(bb_min_latitude.data()), (double *)thrust::raw_pointer_cast(bb_max_latitude.data()), (double *)thrust::raw_pointer_cast(bb_min_longitude.data()), (double *)thrust::raw_pointer_cast(bb_max_longitude.data()), (int *)thrust::raw_pointer_cast(bb_month.data()));
    thrust::for_each(begin, begin + NUMBER_OF_BOUNDING_BOXES, bounding_boxes);
    
    get_events fill_bb((double *)thrust::raw_pointer_cast(latitude.data()), (double *)thrust::raw_pointer_cast(longitude.data()), (int *)thrust::raw_pointer_cast(month.data()), (char *)thrust::raw_pointer_cast(event.data()), (double *)thrust::raw_pointer_cast(bb_min_latitude.data()), (double *)thrust::raw_pointer_cast(bb_max_latitude.data()), (double *)thrust::raw_pointer_cast(bb_min_longitude.data()), (double *)thrust::raw_pointer_cast(bb_max_longitude.data()), (int *)thrust::raw_pointer_cast(bb_month.data()));
    bb_events init;
    init.initialize();
    combine_events add_bb;
    bb_events events = thrust::transform_reduce(begin, begin + linecnt, fill_bb, init, add_bb);
    thrust::device_vector<int> events_count(NUMBER_OF_BOUNDING_BOXES);
    for(int i = 0; i < NUMBER_OF_BOUNDING_BOXES; i++)
        events_count[i] = events.events[i];
    
    //Setup for output
    thrust::device_vector<char> min_latitude_text(NUMBER_OF_BOUNDING_BOXES * LATITUDE_WIDTH);
    thrust::fill(min_latitude_text.begin(), min_latitude_text.end(), 0);
    thrust::device_vector<char> max_latitude_text(NUMBER_OF_BOUNDING_BOXES * LATITUDE_WIDTH);
    thrust::fill(max_latitude_text.begin(), max_latitude_text.end(), 0);
    thrust::device_vector<char> min_longitude_text(NUMBER_OF_BOUNDING_BOXES * LONGITUDE_WIDTH);
    thrust::fill(min_longitude_text.begin(), min_longitude_text.end(), 0);
    thrust::device_vector<char> max_longitude_text(NUMBER_OF_BOUNDING_BOXES * LONGITUDE_WIDTH);
    thrust::fill(max_longitude_text.begin(), max_longitude_text.end(), 0);
    thrust::device_vector<char> month_text(NUMBER_OF_BOUNDING_BOXES * 2);
    thrust::fill(month_text.begin(), month_text.end(), 0);
    thrust::device_vector<char> events_text(NUMBER_OF_BOUNDING_BOXES * EVENT_COUNT_WIDTH);
    thrust::fill(events_text.begin(), events_text.end(), 0);
    thrust::device_vector<int> output_num_columns(1);
    output_num_columns[0] = NUMBER_OF_OUTPUT_COLUMNS;
    thrust::device_vector<char *> output_columns(output_num_columns[0]);
    output_columns[0] = thrust::raw_pointer_cast(min_latitude_text.data());
    output_columns[1] = thrust::raw_pointer_cast(max_latitude_text.data());
    output_columns[2] = thrust::raw_pointer_cast(min_longitude_text.data());
    output_columns[3] = thrust::raw_pointer_cast(min_longitude_text.data());
    output_columns[4] = thrust::raw_pointer_cast(month_text.data());
    output_columns[5] = thrust::raw_pointer_cast(events_text.data());
    thrust::device_vector<int> output_column_width(output_num_columns[0]);
    output_column_width[0] = LATITUDE_WIDTH;
    output_column_width[1] = LATITUDE_WIDTH;
    output_column_width[2] = LONGITUDE_WIDTH;
    output_column_width[3] = LONGITUDE_WIDTH;
    output_column_width[4] = MONTH_WIDTH;
    output_column_width[5] = EVENT_COUNT_WIDTH;
    gpu_ftoa min_lat((double *)thrust::raw_pointer_cast(bb_min_latitude.data()), (char *)thrust::raw_pointer_cast(min_latitude_text.data()), (int *)thrust::raw_pointer_cast(&output_column_width[0]));
    thrust::for_each(begin, begin + NUMBER_OF_BOUNDING_BOXES, min_lat);
    gpu_ftoa max_lat((double *)thrust::raw_pointer_cast(bb_max_latitude.data()), (char *)thrust::raw_pointer_cast(max_latitude_text.data()), (int *)thrust::raw_pointer_cast(&output_column_width[1]));
    thrust::for_each(begin, begin + NUMBER_OF_BOUNDING_BOXES, max_lat);
    gpu_ftoa min_long((double *)thrust::raw_pointer_cast(bb_min_longitude.data()), (char *)thrust::raw_pointer_cast(min_longitude_text.data()), (int *)thrust::raw_pointer_cast(&output_column_width[2]));
    thrust::for_each(begin, begin + NUMBER_OF_BOUNDING_BOXES, min_long);
    gpu_ftoa max_long((double *)thrust::raw_pointer_cast(bb_max_longitude.data()), (char *)thrust::raw_pointer_cast(max_longitude_text.data()), (int *)thrust::raw_pointer_cast(&output_column_width[3]));
    thrust::for_each(begin, begin + NUMBER_OF_BOUNDING_BOXES, max_long);
    gpu_itoa month_totext((int *)thrust::raw_pointer_cast(bb_month.data()), (char *)thrust::raw_pointer_cast(month_text.data()), (int *)thrust::raw_pointer_cast(&output_column_width[4]));
    thrust::for_each(begin, begin + NUMBER_OF_BOUNDING_BOXES, month_totext);
    gpu_itoa events_totext((int *)thrust::raw_pointer_cast(events_count.data()), (char *)thrust::raw_pointer_cast(events_text.data()), (int *)thrust::raw_pointer_cast(&output_column_width[5]));
    thrust::for_each(begin, begin + NUMBER_OF_BOUNDING_BOXES, events_totext);
    
    thrust::copy(events_text.begin(), events_text.end(), ostream_iterator<char>(cout));
    
    thrust::device_vector<int> output_size(1);
    output_size[0] = output_column_width[0] + output_column_width[1] + output_column_width[2] + output_column_width[3] + output_column_width[4] + output_column_width[5] + output_num_columns[0];
    thrust::device_vector<char> output(NUMBER_OF_BOUNDING_BOXES * output_size[0]);
    thrust::fill(output.begin(), output.end(), 0);
    gpu_output format_output((char **)thrust::raw_pointer_cast(output_columns.data()), (char *)thrust::raw_pointer_cast(output.data()), (int *)thrust::raw_pointer_cast(output_column_width.data()), (int *)thrust::raw_pointer_cast(output_size.data()), (int *)thrust::raw_pointer_cast(output_num_columns.data()));
    thrust::for_each(begin, begin + NUMBER_OF_BOUNDING_BOXES, format_output);
    thrust::device_vector<char> final_output(output.size());
    thrust::copy_if(output.begin(), output.end(), final_output.begin(), null_space());
    thrust::device_vector<char>::iterator output_end = thrust::find(final_output.begin(), final_output.end(), NULL);
    
    //Output cleaned data
    HANDLE outputFile = CreateFileA("training_set.csv", GENERIC_WRITE | GENERIC_READ, FILE_SHARE_WRITE, NULL, CREATE_ALWAYS, FILE_ATTRIBUTE_NORMAL | FILE_FLAG_SEQUENTIAL_SCAN, NULL);
    assert(outputFile != INVALID_HANDLE_VALUE);

    HANDLE outputMap = CreateFileMapping(outputFile, NULL, PAGE_READWRITE, 0, output_end - final_output.begin(), NULL);
    if(outputMap == NULL)
        cout << GetLastError();        

    LPVOID outputMapView = MapViewOfFile(outputMap, FILE_MAP_WRITE, 0, 0, 0);
    
    if (outputMapView == NULL)
        cout << GetLastError();

    char *outputMapViewChar = (char *)outputMapView;
    thrust::copy(final_output.begin(), output_end, outputMapViewChar);

    cout << "Output written: " << ((clock() - start)/(double)CLOCKS_PER_SEC) << '\n';
    
    return 0;
}